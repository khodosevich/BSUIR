#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <chrono>

// Вспомогательная функция для проверки ошибок CUDA
void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error (" << msg << "): " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Ядро CUDA для инициализации матрицы случайными значениями
__global__ void init_random_matrix(int* matrix, int rows, int cols, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Проверка границ массива
    if (idx < cols && idy < rows) {
        // Создание генератора CURAND
        hiprandState state;
        hiprand_init(seed, idy * cols + idx, 0, &state); // Инициализация генератора

        // Генерация случайного числа
        matrix[idy * cols + idx] = hiprand(&state) % 101; // Случайное значение от 0 до 100
    }
}

// Ядро CUDA для перестановки элементов с шагом N/2 и M/2
__global__ void rearrange_gpu(const int* matrix, int* result, int rows, int cols) {
    int half_rows = rows / 2;
    int half_cols = cols / 2;

    int i = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    int j = (blockIdx.x * blockDim.x + threadIdx.x) * 2;

    if (i + 1 < rows && j + 1 < cols) {
        // Перестановка элементов из блока 2x2
        result[(i / 2) * cols + (j / 2)] = matrix[i * cols + j];
        result[(i / 2) * cols + (j / 2 + half_cols)] = matrix[i * cols + j + 1];
        result[((i / 2) + half_rows) * cols + (j / 2)] = matrix[(i + 1) * cols + j];
        result[((i / 2) + half_rows) * cols + (j / 2 + half_cols)] = matrix[(i + 1) * cols + j + 1];
    }
}

// Функция для замера времени выполнения на CPU
void measure_cpu(const int* matrix, std::vector<int>& result, int rows, int cols) {
    auto start = std::chrono::high_resolution_clock::now();

    int half_rows = rows / 2;
    int half_cols = cols / 2;

    // Перестановка элементов с шагом N/2 и M/2 на CPU
    for (int i = 0; i < half_rows * 2; i += 2) {
        for (int j = 0; j < half_cols * 2; j += 2) {
            result[i / 2 * cols + j / 2] = matrix[i * cols + j];
            result[i / 2 * cols + (j / 2 + half_cols)] = matrix[i * cols + j + 1];
            result[(i / 2 + half_rows) * cols + j / 2] = matrix[(i + 1) * cols + j];
            result[(i / 2 + half_rows) * cols + (j / 2 + half_cols)] = matrix[(i + 1) * cols + j + 1];
        }
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;

    std::cout << "\nCPU Time: " << duration.count() * 1000 << " milliseconds" << std::endl; // вывод времени в миллисекундах
}

// Функция для замера времени выполнения на GPU
void measure_gpu(const int* d_matrix, int* d_result_matrix, int rows, int cols) {
    int threads_per_block = 32;
    dim3 threads(threads_per_block, threads_per_block);
    dim3 blocks((cols + threads_per_block * 2 - 1) / (threads_per_block * 2),
        (rows + threads_per_block * 2 - 1) / (threads_per_block * 2));

    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate (start)");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate (stop)");

    checkCudaError(hipEventRecord(start), "hipEventRecord (start)");

    // Запуск CUDA ядра
    rearrange_gpu << <blocks, threads >> > (d_matrix, d_result_matrix, rows, cols);
    checkCudaError(hipGetLastError(), "Kernel execution"); // Проверка на ошибку ядра
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize");

    checkCudaError(hipEventRecord(stop), "hipEventRecord (stop)");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize");

    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop), "hipEventElapsedTime");
    std::cout << "\nGPU Time: " << milliseconds << " milliseconds" << std::endl; // вывод времени в миллисекундах

    checkCudaError(hipEventDestroy(start), "hipEventDestroy (start)");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy (stop)");
}

// Полное поэлементное сравнение матриц
bool compare_results(const std::vector<int>& cpu_matrix, const std::vector<int>& gpu_matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            if (cpu_matrix[i * cols + j] != gpu_matrix[i * cols + j]) { // Проверка на равенство
                return false;
            }
        }
    }
    return true;
}

int main() {
    int rows = 5000, cols = 5000;
    size_t matrix_size = rows * cols * sizeof(int);

    int* d_matrix;
    int* d_matrix_res;
    checkCudaError(hipMalloc(&d_matrix, matrix_size), "hipMalloc (d_matrix)");
    checkCudaError(hipMalloc(&d_matrix_res, matrix_size), "hipMalloc (d_matrix_res)");

    // Инициализация матрицы случайными значениями на GPU
    dim3 threads(32, 32);
    dim3 blocks((cols + threads.x - 1) / threads.x, (rows + threads.y - 1) / threads.y);
    unsigned long long seed = 1234; // Уникальный сид для генерации случайных чисел
    init_random_matrix << <blocks, threads >> > (d_matrix, rows, cols, seed);
    checkCudaError(hipGetLastError(), "Kernel execution for init_random_matrix");
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize for init_random_matrix");

    // Копирование данных из GPU в CPU для последующей проверки
    std::vector<int> cpu_matrix(rows * cols);
    checkCudaError(hipMemcpy(cpu_matrix.data(), d_matrix, matrix_size, hipMemcpyDeviceToHost), "hipMemcpy DeviceToHost");

    // Результат на CPU
    std::vector<int> cpu_result(rows * cols);
    measure_cpu(cpu_matrix.data(), cpu_result, rows, cols);

    // Запуск на GPU
    measure_gpu(d_matrix, d_matrix_res, rows, cols);

    // Получение результата с GPU
    std::vector<int> gpu_result(rows * cols);
    checkCudaError(hipMemcpy(gpu_result.data(), d_matrix_res, matrix_size, hipMemcpyDeviceToHost), "hipMemcpy DeviceToHost");

    // Сравнение результатов
    if (compare_results(cpu_result, gpu_result, rows, cols)) {
        std::cout << "CPU and GPU results match!" << std::endl;
    }
    else {
        std::cout << "Results differ!" << std::endl;
    }

    std::cout << "\nPartial Matrix CPU:" << std::endl;
    for (int i = 0; i < 10; ++i) {
        for (int j = 0; j < 10; ++j) {
            std::cout << cpu_result[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "\nPartial Matrix GPU:" << std::endl;
    for (int i = 0; i < 10; ++i) {
        for (int j = 0; j < 10; ++j) {
            std::cout << gpu_result[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }

    // Освобождение памяти
    checkCudaError(hipFree(d_matrix), "hipFree (d_matrix)");
    checkCudaError(hipFree(d_matrix_res), "hipFree (d_matrix_res)");

    return 0;
}
