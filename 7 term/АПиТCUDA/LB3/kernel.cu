#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <cmath>

#define BLOCK_SIZE 32
#define GRID_SIZE 256

#define DATA_PACK_SIZE BLOCK_SIZE*BLOCK_SIZE*GRID_SIZE*GRID_SIZE

#define CHECK_CUDA(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        printf("CUDA Error: %s:%d, %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CURAND(call) { \
    const hiprandStatus_t error = call; \
    if (error != HIPRAND_STATUS_SUCCESS) { \
        printf("CURAND Error: %s:%d, %d\n", __FILE__, __LINE__, error); \
        exit(EXIT_FAILURE); \
    } \
}

using data_t = int;

struct dataPack
{
    data_t data[DATA_PACK_SIZE]{};
};

enum class ReductionOp {
    SUM,
    XOR,
    AND,
    OR,
    MIN,
    MAX,
    SUB
};

template<ReductionOp Op>
struct Reducer {
    __host__ __device__ static void reduce(data_t* a, const data_t& b);

    __device__ static void reduceAtomic(data_t* a, const data_t& b);
};

template<>
struct Reducer<ReductionOp::SUM> {
    __host__ __device__ static void reduce(data_t* a, const data_t& b) { *a += b; }

    __device__ static void reduceAtomic(data_t* a, const data_t& b) { atomicAdd(a, b); }
};

template<>
struct Reducer<ReductionOp::XOR> {
    __host__ __device__ static void reduce(data_t* a, const data_t& b) { *a ^= b; }
    __device__ static void reduceAtomic(data_t* a, const data_t& b) { atomicXor(a, b); }
};

template<>
struct Reducer<ReductionOp::AND> {
    __host__ __device__ static void reduce(data_t* a, const data_t& b) { *a &= b; }
    __device__ static void reduceAtomic(data_t* a, const data_t& b) { atomicAnd(a, b); }
};

template<>
struct Reducer<ReductionOp::OR> {
    __host__ __device__ static void reduce(data_t* a, const data_t& b) { *a |= b; }
    __device__ static void reduceAtomic(data_t* a, const data_t& b) { atomicOr(a, b); }
};

template<>
struct Reducer<ReductionOp::MIN> {
    __host__ __device__ static void reduce(data_t* a, const data_t& b) { *a = *a < b ? *a : b; }
    __device__ static void reduceAtomic(data_t* a, const data_t& b) { atomicMin(a, b); }
};

template<>
struct Reducer<ReductionOp::MAX> {
    __host__ __device__ static void reduce(data_t* a, const data_t& b) { *a = *a > b ? *a : b; }
    __device__ static void reduceAtomic(data_t* a, const data_t& b) { atomicMax(a, b); }
};

template<>
struct Reducer<ReductionOp::SUB> {
    __host__ __device__ static void reduce(data_t* a, const data_t& b) { *a -= b; }
    __device__ static void reduceAtomic(data_t* a, const data_t& b) { atomicSub(a, b); }
};

template<ReductionOp Op>
data_t processMatrixCPU(const dataPack& data) {
    data_t output{};
    for (int globalY = 0; globalY < BLOCK_SIZE * GRID_SIZE; globalY++) {
        for (int globalX = 0; globalX < BLOCK_SIZE * GRID_SIZE; globalX++) {
            int globalIdx = globalY * BLOCK_SIZE * GRID_SIZE + globalX;
            Reducer<Op>::reduce(&output, data.data[globalIdx]);
        }
    }
    return output;
}

template<ReductionOp Op>
__global__ void processMatrix(dataPack* input, data_t* result) {
    __shared__ data_t sharedSum[BLOCK_SIZE * BLOCK_SIZE];

    int localX = threadIdx.x;
    int localY = threadIdx.y;
    int blockX = blockIdx.x;
    int blockY = blockIdx.y;

    int globalX = blockX * blockDim.x + localX;
    int globalY = blockY * blockDim.y + localY;

    int localIdx = localY * blockDim.x + localX;
    int globalIdx = globalY * blockDim.x * gridDim.x + globalX;

    sharedSum[localIdx] = input->data[globalIdx];

    __syncthreads();

    for (unsigned int stride = blockDim.x * blockDim.y / 2; stride > 0; stride >>= 1) {
        if (localIdx < stride) {
            Reducer<Op>::reduce(sharedSum + localIdx, sharedSum[localIdx + stride]);
        }
        __syncthreads();
    }

    if (localIdx == 0) {
        Reducer<Op>::reduceAtomic(result, sharedSum[0]);
    }
}


template<ReductionOp Op>
void testReduction(const char* opName) {
    static dataPack inputData;
    dataPack* deviceInputData;
    data_t* deviceResult;

    // Allocate device memory
    CHECK_CUDA(hipMalloc(&deviceInputData, sizeof(dataPack)));
    CHECK_CUDA(hipMalloc(&deviceResult, sizeof(data_t)));

    // Generate random data
    hiprandGenerator_t gen;
    CHECK_CURAND(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    CHECK_CURAND(hiprandGenerate(gen, reinterpret_cast<unsigned int*>(deviceInputData), DATA_PACK_SIZE));
    CHECK_CUDA(hipMemcpy(&inputData, deviceInputData, sizeof(dataPack), hipMemcpyDeviceToHost));

    // Create events for timing
    hipEvent_t start_gpu_total, stop_gpu_total, start_gpu, stop_gpu, start_cpu, stop_cpu;
    CHECK_CUDA(hipEventCreate(&start_gpu_total));
    CHECK_CUDA(hipEventCreate(&stop_gpu_total));
    CHECK_CUDA(hipEventCreate(&start_gpu));
    CHECK_CUDA(hipEventCreate(&stop_gpu));
    CHECK_CUDA(hipEventCreate(&start_cpu));
    CHECK_CUDA(hipEventCreate(&stop_cpu));

    // Total GPU computation including data transfer
    CHECK_CUDA(hipEventRecord(start_gpu_total));

    // Copy data to device
    CHECK_CUDA(hipMemcpy(deviceInputData, &inputData, sizeof(dataPack), hipMemcpyHostToDevice));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(GRID_SIZE, GRID_SIZE);

    // GPU computation
    CHECK_CUDA(hipEventRecord(start_gpu));
    processMatrix<Op> << <dimGrid, dimBlock >> > (deviceInputData, deviceResult);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipEventRecord(stop_gpu));

    data_t gpuResult;
    CHECK_CUDA(hipMemcpy(&gpuResult, deviceResult, sizeof(data_t), hipMemcpyDeviceToHost));

    // Stop total GPU timing
    CHECK_CUDA(hipEventRecord(stop_gpu_total));

    // CPU computation
    CHECK_CUDA(hipEventRecord(start_cpu));
    data_t cpuResult = processMatrixCPU<Op>(inputData);
    CHECK_CUDA(hipEventRecord(stop_cpu));

    // Get timing results
    float gpu_milliseconds = 0;
    float gpu_total_milliseconds = 0;
    float cpu_milliseconds = 0;
    CHECK_CUDA(hipEventSynchronize(stop_gpu));
    CHECK_CUDA(hipEventSynchronize(stop_gpu_total));
    CHECK_CUDA(hipEventSynchronize(stop_cpu));
    CHECK_CUDA(hipEventElapsedTime(&gpu_milliseconds, start_gpu, stop_gpu));
    CHECK_CUDA(hipEventElapsedTime(&gpu_total_milliseconds, start_gpu_total, stop_gpu_total));
    CHECK_CUDA(hipEventElapsedTime(&cpu_milliseconds, start_cpu, stop_cpu));

    // Calculate speedups
    float speedup_excluding_data_transfer = cpu_milliseconds / gpu_milliseconds;
    float speedup_including_data_transfer = cpu_milliseconds / gpu_total_milliseconds;

    // Print results
    printf("\n=== Testing %s Operation ===\n", opName);
    printf("GPU Time (Excluding Data Transfer): %f ms\n", gpu_milliseconds);
    printf("GPU Time (Including Data Transfer): %f ms\n", gpu_total_milliseconds);
    printf("CPU Time: %f ms\n", cpu_milliseconds);
    printf("Speedup Excluding Data Transfer: %fx\n", speedup_excluding_data_transfer);
    printf("Speedup Including Data Transfer: %fx\n", speedup_including_data_transfer);
    printf("Result (GPU): %d\n", gpuResult);
    printf("Result (CPU): %d\n", cpuResult);
    printf("Results %s\n", (gpuResult == cpuResult) ? "match" : "don't match");
    printf("============================\n");

    // Cleanup
    CHECK_CUDA(hipEventDestroy(start_gpu_total));
    CHECK_CUDA(hipEventDestroy(stop_gpu_total));
    CHECK_CUDA(hipEventDestroy(start_gpu));
    CHECK_CUDA(hipEventDestroy(stop_gpu));
    CHECK_CUDA(hipEventDestroy(start_cpu));
    CHECK_CUDA(hipEventDestroy(stop_cpu));
    CHECK_CUDA(hipFree(deviceInputData));
    CHECK_CUDA(hipFree(deviceResult));
    CHECK_CURAND(hiprandDestroyGenerator(gen));
}

int main() {
    testReduction<ReductionOp::SUM>("SUM");
    testReduction<ReductionOp::XOR>("XOR");
    testReduction<ReductionOp::AND>("AND");
    testReduction<ReductionOp::OR>("OR");
    testReduction<ReductionOp::MIN>("MIN");
    testReduction<ReductionOp::MAX>("MAX");
    testReduction<ReductionOp::SUB>("SUB");

    return 0;
}
