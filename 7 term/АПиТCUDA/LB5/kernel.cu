#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include ""
#include <chrono>
#include <iostream>
#include <vector>
#include <opencv2/opencv.hpp>
#include <opencv2/imgproc.hpp>

#define MASK_WIDTH 5
#define MASK_RADIUS (MASK_WIDTH / 2)
#define THREADS_PER_BLOCK 32

template <typename T>
T clamp(T val, T minVal, T maxVal) {
    return (val < minVal) ? minVal : (val > maxVal ? maxVal : val);
}

bool loadImage(const std::string& filename, int& width, int& height, std::vector<unsigned char>& data, bool& isGrayscale) {
    cv::Mat img = cv::imread(filename, cv::IMREAD_UNCHANGED);
    if (img.empty()) return false;

    isGrayscale = (img.channels() == 1);
    width = img.cols;
    height = img.rows;

    if (!isGrayscale) {
        cv::cvtColor(img, img, cv::COLOR_BGR2RGB);
    }

    data.assign(img.data, img.data + img.total() * img.elemSize());
    return true;
}

bool saveImage(const std::string& filename, int width, int height, const std::vector<unsigned char>& data, bool isGrayscale) {

    cv::Mat img(height, width, isGrayscale ? CV_8UC1 : CV_8UC3, const_cast<unsigned char*>(data.data()));
    if (!isGrayscale) {
        cv::cvtColor(img, img, cv::COLOR_RGB2BGR);
    }
    return cv::imwrite(filename, img);
}

void applyMinFilterCPU(const std::vector<unsigned char>& input, std::vector<unsigned char>& output, int width, int height) {
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            for (int c = 0; c < 3; ++c) {
                int min_val = 255;

                for (int dy = -MASK_RADIUS; dy <= MASK_RADIUS; ++dy) {
                    for (int dx = -MASK_RADIUS; dx <= MASK_RADIUS; ++dx) {
                        int ny = clamp(y + dy, 0, height - 1);
                        int nx = clamp(x + dx, 0, width - 1);
                        int pixel = input[(ny * width + nx) * 3 + c];
                        min_val = std::min(min_val, pixel);
                    }
                }

                output[(y * width + x) * 3 + c] = static_cast<unsigned char>(min_val);
            }
        }
    }
}

void applyMinFilterCPU_Grayscale(const std::vector<unsigned char>& input, std::vector<unsigned char>& output, int width, int height) {
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            int min_val = 255;

            for (int dy = -MASK_RADIUS; dy <= MASK_RADIUS; ++dy) {
                for (int dx = -MASK_RADIUS; dx <= MASK_RADIUS; ++dx) {
                    int ny = clamp(y + dy, 0, height - 1);
                    int nx = clamp(x + dx, 0, width - 1);
                    int pixel = input[ny * width + nx];
                    min_val = std::min(min_val, pixel);
                }
            }

            output[y * width + x] = static_cast<unsigned char>(min_val);
        }
    }
}

__global__ void applyMinFilterCUDA(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        for (int c = 0; c < 3; ++c) {
            int min_val = 255;

            for (int dy = -MASK_RADIUS; dy <= MASK_RADIUS; ++dy) {
                for (int dx = -MASK_RADIUS; dx <= MASK_RADIUS; ++dx) {
                    int nx = min(max(x + dx, 0), width - 1);
                    int ny = min(max(y + dy, 0), height - 1);
                    int pixel = input[(ny * width + nx) * 3 + c];
                    min_val = min(min_val, pixel);
                }
            }

            output[(y * width + x) * 3 + c] = min_val;
        }
    }
}

__global__ void applyMinFilterCUDA_Grayscale(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int min_val = 255;

        for (int dy = -MASK_RADIUS; dy <= MASK_RADIUS; ++dy) {
            for (int dx = -MASK_RADIUS; dx <= MASK_RADIUS; ++dx) {
                int nx = min(max(x + dx, 0), width - 1);
                int ny = min(max(y + dy, 0), height - 1);
                int pixel = input[ny * width + nx];
                min_val = min(min_val, pixel);
            }
        }

        output[y * width + x] = min_val;
    }
}

int main() {
    int width, height;
    bool isGrayscale;
    std::string path = "2.jpg";
    std::vector<unsigned char> image;

    if (!loadImage(path, width, height, image, isGrayscale)) {
        std::cerr << "Failed to load image!" << std::endl;
        return -1;
    }

    std::vector<unsigned char> outputCPU(image.size());
    std::vector<unsigned char> outputGPU(image.size());

    auto startCPU = std::chrono::high_resolution_clock::now();
    if (isGrayscale) {
        applyMinFilterCPU_Grayscale(image, outputCPU, width, height);
    }
    else {
        applyMinFilterCPU(image, outputCPU, width, height);
    }
    auto endCPU = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> durationCPU = endCPU - startCPU;
    std::cout << "CPU time: " << durationCPU.count() << " seconds\n";

    unsigned char* d_input, * d_output;
    hipMalloc(&d_input, image.size());
    hipMalloc(&d_output, image.size());

    auto startGPUcpy = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_input, image.data(), image.size(), hipMemcpyHostToDevice);

    dim3 blockSize(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 gridSize((width + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, (height + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    auto startGPU = std::chrono::high_resolution_clock::now();
    if (isGrayscale) {
        applyMinFilterCUDA_Grayscale << <gridSize, blockSize >> > (d_input, d_output, width, height);
    }
    else {
        applyMinFilterCUDA << <gridSize, blockSize >> > (d_input, d_output, width, height);
    }
    hipDeviceSynchronize();
    auto endGPU = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> durationGPU = endGPU - startGPU;
    std::cout << "GPU time: " << durationGPU.count() << " seconds\n";

    hipMemcpy(outputGPU.data(), d_output, image.size(), hipMemcpyDeviceToHost);

    auto endGPUcpy = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> durationGPUcpy = endGPUcpy - startGPUcpy;
    std::cout << "GPU memcpy time: " << durationGPUcpy.count() << " seconds\n";

    bool match = true;
    for (size_t i = 0; i < image.size(); ++i) {
        if (outputCPU[i] != outputGPU[i]) {
            match = false;
            std::cout << "Mismatch at pixel " << i << ": CPU = " << static_cast<int>(outputCPU[i]) << ", GPU = " << static_cast<int>(outputGPU[i]) << "\n";
            break;
        }
    }
    if (match) {
        std::cout << "CPU and GPU results match.\n";
    }
    else {
        std::cout << "CPU and GPU results do not match.\n";
    }

    saveImage("outputCPU.jpg", width, height, outputCPU, isGrayscale);
    saveImage("outputGPU.jpg", width, height, outputGPU, isGrayscale);

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
